#include "hip/hip_runtime.h"
#include "cnn/hip/hip_runtime.h"
#include "cnn/gpu-ops.h"
#include "cnn/gpu-kernels.h"
#include "cnn/functors.h"
#include <thrust/version.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

namespace cnn {
namespace gpu {

// this wraps kernel dispatches for various operations (preventing us from
// having to compile a version of nodes.cc with NVCC)

    void saxpy_fast(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
    {
        // Y <- A * X + Y
        thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy_functor(A));
    }

    void add_to(int n, const float* x, float *y)
    {
        thrust::device_ptr<float> src_ptr = thrust::device_pointer_cast((float*)x);
        thrust::device_ptr<float> tgt_ptr = thrust::device_pointer_cast(y);
        // Y <- A * X + Y
        thrust::transform(src_ptr, src_ptr + n, tgt_ptr, tgt_ptr, thrust::plus<float>()); 
    }

    void set_to_value_of(int n, float* x0, float val) {
        thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(x0);
        thrust::fill(thrust::device, dev_ptr, dev_ptr + n, val);
    }

void set_to_value_of(int n, float* x0, float *val) {
    thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(x0);
    thrust::device_ptr<float> src_dev_ptr = thrust::device_pointer_cast(val);
    thrust::copy(src_dev_ptr, src_dev_ptr + n, dev_ptr);
}

void vpairwise_rank_loss(int n, float margin, const float* xgood, const float* xbad, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  binaryExprKernel<<<tb.first, tb.second>>>(n, xgood, xbad, y, FPairwiseRankLoss(margin));
}

void vpairwise_rank_loss_backward(int n, bool d_wrt_correct, const float* fx, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  if (d_wrt_correct) {
    accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FRectifyNegateBackward());
  } else {
    accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FRectifyBackward());
  }
}

void vcwise_product(int n, const float* x0, const float* x1, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  binaryExprKernel<<<tb.first, tb.second>>>(n, x0, x1, y, FProduct());
}

void vcwise_product_backward(int n, const float* dEdy, const float* x_other, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, dEdy, x_other, dEdx, FProduct());
}

void vcwise_quotient(int n, const float* x0, const float* x1, float* y) {
    auto tb = SizeToBlockThreadPair(n);
    binaryExprKernel << <tb.first, tb.second >> >(n, x0, x1, y, FQuotient());
}

void vcwise_quotient_backward(int n, const float* dEdy, const float* x_other, float* dEdx) {
    auto tb = SizeToBlockThreadPair(n);
    accBinaryExprKernel << <tb.first, tb.second >> >(n, dEdy, x_other, dEdx, FQuotient());
}

void vconstant_minusx(int n, float c, const float* x, float* y) {
    auto tb = SizeToBlockThreadPair(n);
    unaryExprKernel << <tb.first, tb.second >> >(n, x, y, FConstantMinus(c));
}

void vconstant_multiplyx(int n, float c, const float* x, float* y) {
    auto tb = SizeToBlockThreadPair(n);
    unaryExprKernel << <tb.first, tb.second >> >(n, x, y, FConstantMultiply(c));
}

void vconstant_multiplyx_backward(int n, float c, const float* x, float* y) {
    auto tb = SizeToBlockThreadPair(n);
    accUnaryExprKernel << <tb.first, tb.second >> >(n, x, y, FConstantMultiply(c));
}

void vexp(int n, const float* x, float* y) {
    auto tb = SizeToBlockThreadPair(n);
    unaryExprKernel << <tb.first, tb.second >> >(n, x, y, FExp());
}

void vlog(int n, const float* x, float* y) {
    auto tb = SizeToBlockThreadPair(n);
    unaryExprKernel << <tb.first, tb.second >> >(n, x, y, FLog());
}

void vnegate(int n, const float* x, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  unaryExprKernel<<<tb.first, tb.second>>>(n, x, y, FNegate());
}

void vnegate_backward(int n, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accUnaryExprKernel<<<tb.first, tb.second>>>(n, dEdf, dEdx, FNegate());
}

void vrelu(int n, const float* x, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  unaryExprKernel<<<tb.first, tb.second>>>(n, x, y, FRectify());
}

void vrelu_backward(int n, const float* fx, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FRectifyBackward());
}

void vtanh(int n, const float* x, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  unaryExprKernel<<<tb.first, tb.second>>>(n, x, y, FTanh());
}

void vtanh_backward(int n, const float* fx, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FTanhBackward());
}

void vlogistic(int n, const float* x, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  unaryExprKernel<<<tb.first, tb.second>>>(n, x, y, FLogisticSigmoid());
}

void vlogistic_backward(int n, const float* fx, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FLogisticSigmoidBackward());
}

void sqeucdist_backward(int n, const float* dEdy, const float* x0, const float* x1, float* dEdx, int i) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, x0, x1, dEdx, FEuclideanBackward(i, dEdy));
}

void sgd_update(int n, const float* g, float* x, float scale, float lambda) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, x, g, x, FL2SGDUpdate(lambda, scale));
}

// adapted from NVIDIA example
__global__ void ker_sqeucdist(int n, const float *x0, const float *x1, float* res) {
  __shared__ float buf[256];
  for (int i = threadIdx.x; i < 256; i += blockDim.x) {
    float sum = 0;
    for (int pos = i; pos < n; pos += 256) {
      const float d = x0[pos] - x1[pos];
      sum += d * d;
    }
    buf[i] = sum;
  }
  for (int stride = 128; stride > 0; stride >>= 1) {
    __syncthreads();
    for (int i = threadIdx.x; i < stride; i += blockDim.x)
        buf[i] += buf[stride + i];
  }
  __syncthreads();
  if (threadIdx.x == 0) res[0] = buf[0];
}

void sqeucdist(int n, const float* x0, const float *x1, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  ker_sqeucdist<<<tb.first,tb.second>>>(n, x0, x1, y);
}

// adapted from NVIDIA example
__global__ void ker_l2_norm_reducer(int n, const float *x0, float* res, bool sq, bool acc) {
  __shared__ float buf[256];
  for (int i = threadIdx.x; i < 256; i += blockDim.x) {
    float sum = 0;
    for (int pos = i; pos < n; pos += 256) {
      const float d = x0[pos];
      sum += sq ? d * d : d;
    }
    buf[i] = sum;
  }
  for (int stride = 128; stride > 0; stride >>= 1) {
    __syncthreads();
    for (int i = threadIdx.x; i < stride; i += blockDim.x)
        buf[i] += buf[stride + i];
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    if (acc) res[0] += buf[0]; else res[0] = buf[0];
  }
}

void l2_norm_reducer(int n, const float* x0, float* y, bool square, bool accumulate) {
  auto tb = SizeToBlockThreadPair(n);
  ker_l2_norm_reducer<<<tb.first,tb.second>>>(n, x0, y, square, accumulate);
}

// adapted from NVIDIA example
__global__ void ker_softmax(int n, const float *x0, float* res) {
  __shared__ float buf[256];
  for (int i = threadIdx.x; i < 256; i += blockDim.x) {
    float me = __int_as_float(0xff800000);
    for (int pos = i; pos < n; pos += 256) {
      const float d = x0[pos];
      me = d > me ? d : me;
    }
    buf[i] = me;
  }
  for (int stride = 128; stride > 0; stride >>= 1) {
    __syncthreads();
    for (int i = threadIdx.x; i < stride; i += blockDim.x)
        buf[i] = buf[i] > buf[stride + i] ? buf[i] : buf[stride + i];
  }
  __syncthreads();
  const float max_elem = buf[0];
  for (int i = threadIdx.x; i < 256; i += blockDim.x) {
    float sum = 0;
    for (int pos = i; pos < n; pos += 256)
      sum += expf(x0[pos] - max_elem);
    buf[i] = sum;
  }
  for (int stride = 128; stride > 0; stride >>= 1) {
    __syncthreads();
    for (int i = threadIdx.x; i < stride; i += blockDim.x)
        buf[i] += buf[stride + i];
  }
  __syncthreads();
  float lz = log(buf[0]) + max_elem;
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    res[i] = exp(x0[i] - lz);
    i += gridDim.x * blockDim.x;
  }
}

void softmax(int n, const float* x0, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  ker_softmax<<<tb.first,tb.second>>>(n, x0, y);
}

// A kernel to calculate the dot product between two arrays
__global__ void ker_dotproduct(int n, const float* x, const float* y, float* z) {
  __shared__ float buf[256];
  for (int i = threadIdx.x; i < 256; i += blockDim.x) {
    float sum = 0;
    for (int pos = i; pos < n; pos += 256)
      sum += x[pos] * y[pos];
    buf[i] = sum;
  }
  for (int stride = 128; stride > 0; stride >>= 1) {
    __syncthreads();
    for (int i = threadIdx.x; i < stride; i += blockDim.x)
        buf[i] += buf[stride + i];
  }
  __syncthreads();
  if (threadIdx.x == 0)
    z[0] = buf[0];
}

void softmax_backward(int n, const float* fx, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  float* gpu_ods;
  float ods;
  hipMalloc((void **)&gpu_ods, sizeof(float));
  ker_dotproduct<<<tb.first, tb.second>>>(n, fx, dEdf, gpu_ods);
  hipMemcpy(&ods, gpu_ods, sizeof(float), hipMemcpyDeviceToHost);
  hipFree(gpu_ods);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FSoftmaxBackward(-ods));
}

void logsoftmax_backward(int n, const float* fx, const float* dEdf, float* dEdx) 
{
    /*
    float off_diag_sum = 0;
    for (auto p : as_vector(dEdf))
    off_diag_sum += p;
    off_diag_sum *= -1;
    *dEdxi += (*fx).binaryExpr(*dEdf, FLogSoftmaxBackward(off_diag_sum));
    */
    thrust::device_ptr<float> dp = thrust::device_pointer_cast((float*)fx);
    thrust::device_ptr<float> de = thrust::device_pointer_cast((float*)dEdf);
    thrust::device_ptr<float> dr = thrust::device_pointer_cast(dEdx);
    thrust::device_vector<float> dtemp(n);
//    thrust::transform(dp, dp + n, de, dtemp.begin(), FWeightedError());
    float off_diag_sum  = - thrust::reduce(de, de + n);
    thrust::transform(dp, dp + n, de, dtemp.begin(), FLogSoftmaxBackward(off_diag_sum)); 
    thrust::transform(dtemp.begin(), dtemp.end(), dr, dr, thrust::plus<float>());
}

// adapted from NVIDIA example
__global__ void ker_pnlsoftmax(int n, int elem_idx, const float *x0, float* res, float* logz) {
  __shared__ float buf[256];
  for (int i = threadIdx.x; i < 256; i += blockDim.x) {
    float me = __int_as_float(0xff800000);
    for (int pos = i; pos < n; pos += 256) {
      const float d = x0[pos];
      me = d > me ? d : me;
    }
    buf[i] = me;
  }
  for (int stride = 128; stride > 0; stride >>= 1) {
    __syncthreads();
    for (int i = threadIdx.x; i < stride; i += blockDim.x)
        buf[i] = buf[i] > buf[stride + i] ? buf[i] : buf[stride + i];
  }
  __syncthreads();
  const float max_elem = buf[0];
  for (int i = threadIdx.x; i < 256; i += blockDim.x) {
    float sum = 0;
    for (int pos = i; pos < n; pos += 256)
      sum += expf(x0[pos] - max_elem);
    buf[i] = sum;
  }
  for (int stride = 128; stride > 0; stride >>= 1) {
    __syncthreads();
    for (int i = threadIdx.x; i < stride; i += blockDim.x)
        buf[i] += buf[stride + i];
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    float lz = log(buf[0]) + max_elem;
    logz[0] = lz;
    res[0] = lz - x0[elem_idx];
  }
}

void pnlsoftmax(int n, int elem_idx, const float* x0, float* y, float* logz) {
  auto tb = SizeToBlockThreadPair(n);
  ker_pnlsoftmax<<<tb.first,tb.second>>>(n, elem_idx, x0, y, logz);
}

__global__ void fixup_pnl(const float* dEdf, float* dEdxi, int i) {
  if (threadIdx.x == 0) dEdxi[i] -= dEdf[0];
}

void pnlsoftmax_backward(int n, int elem_idx, const float* x0, const float* dEdf, const float* logz, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accUnaryExprKernel<<<tb.first, tb.second>>>(n, x0, dEdx, FPtrNegLogSoftmaxBackward(logz, dEdf));
  fixup_pnl<<<1,1>>>(dEdf, dEdx, elem_idx);
}


void conv1dwide(const int n, const int m, const float* xs, const int k, const float *fx, float *fy)
{

    thrust::device_vector<float> dv((m + k) * n, 0.0);
    thrust::device_ptr<float> vp = dv.data();
    thrust::device_ptr<float> fp((float*)fx);
    thrust::device_ptr<float> xp((float*)xs);
    thrust::device_ptr<float> yp(fy);

    for (size_t tk = 0; tk < k; tk++)
    {
        for (size_t j = 0; j < m; j++)
            thrust::transform(xp + j * n, xp + (j + 1) * n, fp + tk * n, vp + tk * n + j * n, thrust::multiplies<float>());
    }
    thrust::copy(vp, vp + (m + k) * n, thrust::device_pointer_cast(fy));
}

void conv1dwide_backward(const int i, const int n, const int m, const float* xs, const int k, const float *fx, const float* dEdf, float *dEdx)
{
    thrust::device_vector<float> dv(m  * n, 0.0);
    thrust::device_ptr<float> vp = dv.data();
    thrust::device_ptr<float> fp((float*)fx);
    thrust::device_ptr<float> xp((float*)xs);
    thrust::device_ptr<float> d((float*)dEdf);
    thrust::device_ptr<float> yp(dEdx);

    for (size_t tk = 0; tk < k; tk++)
    {
        if (i == 0) { // derivative wrt input x
            for (size_t j = 0; j < m; j++)
                thrust::transform(d + j * n + tk*n, d + (j + 1) * n + tk*n, fp + tk * n, dv.data() + j * n, thrust::multiplies<float>());
        }
        else { // derivative wrt filter f
            for (size_t j = 0; j < m; j++)
                thrust::transform(d + j * n + tk*n, d + (j + 1) * n + tk*n, xp + j * n, dv.data() + tk * n, thrust::multiplies<float>());
        }
    }
    if (i == 0)
        thrust::transform(dv.data(), dv.data() + m * n, yp, yp, thrust::plus<float>());
    else 
        thrust::transform(dv.data(), dv.data() + k * n, yp, yp, thrust::plus<float>());
}

void addVectorToAllColumns(const int n, const float * xs, const int m, const float* fx, float *fy)
{
    thrust::device_ptr<float> fp((float*)fx);
    thrust::device_ptr<float> xp((float*)xs);
    thrust::device_ptr<float> yp(fy);
    for (size_t j = 0; j < n / m; j++)
        thrust::transform(xp + j * m, xp + (j + 1) * m, fp, yp + j * m, thrust::plus<float>());
}

void addVectorToAllColumns_backward(const int i, const int r, const int c, const float* dEdf, float *dEdxi)
{
    thrust::device_ptr<const float> dp(dEdf);
    thrust::device_ptr<float> dx(dEdxi);

    if (i == 0)
    {
        // x
        thrust::transform(dp, dp + r * c, dx, dx, thrust::plus<float>());
    }
    else
    {
        // bias
        for (int k = 0; k < c; k++)
            thrust::transform(dp + k * r, dp + (k + 1)*r, dx, dx, thrust::plus<float>());
    }
}

/**
stride : the jump step
*/
void foldRows(const int n, const int m, const float *xs, const int stride, const int orows, float *fy)
{
    thrust::device_ptr<float> xp((float*)xs), pp;
    thrust::device_ptr<float> yp(fy);
    thrust::host_vector<float> vo(orows * m);

    pp = xp;
    for (size_t j = 0; j < m; j++)
    {
        for (size_t r = 0; r < orows; r++)
        {
            vo[j * orows + r] = thrust::reduce(pp, pp + stride);
            pp += stride;
        }
    }
}

void foldRows_backward(const int orows, const float* dEdf, const int n, const int m, float *fy)
{
    thrust::device_ptr<float> dp((float*)dEdf);
    thrust::device_ptr<float> yp(fy);

    for (int i = 0; i < orows; ++i)
    {
        int stride = n / orows;
        for (int j = 0; j < m; j++)
        { // loop over columns
            for (int k = 0; k < stride; k++)
            {
                *(yp + i * stride + k + j * n) += *(dp + i + j * n);
            }
        }
    }
}

void kMaxPooling(const int n, const int m, const float *xs, const int k, float *fy, int* aux_mem)
{
    thrust::device_ptr<float> xp((float*)xs), pp;
    thrust::device_ptr<float> yp(fy);
    thrust::device_vector<float> vo(m);
    thrust::device_vector<float> vp(k);

    pp = xp;

    int* maxmap = static_cast<int*>(aux_mem);
    size_t mi = 0;
    for (unsigned i = 0; i < n; ++i) {
        for (size_t j = 0; j < m; j++)
            vo[j] = (*(pp + i + j * n));
        thrust::sort(thrust::device, vo.data(), vo.data() + m);

        size_t mk = 0;
        for (int j = 0; j < m; j++)
        {
            if (mk == k)
                break;
            if (*(pp + i + j * n) >= vo[m - k])
            {
                *(yp + i + mk * n) = *(pp + i + j*n);
                hipMemcpy(&maxmap[mi], &j, sizeof(int), hipMemcpyHostToDevice); 
                mi++;
                mk++;
            }
        }
    }
}

void kMaxPooling_backward(const int n, const int m, const float *xs, const int k, const float * dEdf, float *dEdxi, const int* aux_mem)
{
    const int* maxmap = aux_mem;
    int mk = 0;
    int oj;
    thrust::device_ptr<const float> xp(xs);
    thrust::device_ptr<const float> dp(dEdf);
    thrust::device_ptr<float> yp(dEdxi);
    thrust::host_vector<int> hv(n, 0);
    hipMemcpy(hv.data(), maxmap, sizeof(int)*n, hipMemcpyDeviceToHost);

    for (unsigned i = 0; i < n; ++i) {
        for (unsigned j = 0; j < k; ++j) {
            oj = hv[mk++];
            if (oj < k && oj >= 0){
                thrust::transform(dp + i + j * n, dp + i + j * n + 1, yp + i + oj * n, yp + i + oj * n, thrust::plus<float>());
            }
        }
    }
}


} // namespace gpu
} // namespace cnn
