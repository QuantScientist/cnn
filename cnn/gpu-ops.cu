#include "hip/hip_runtime.h"
#include "cnn/hip/hip_runtime.h"
#include "cnn/gpu-ops.h"
#include "cnn/gpu-kernels.h"
#include "cnn/functors.h"
#include <thrust/version.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include "gpu-ops.cuh"

namespace cnn {
namespace gpu {

// this wraps kernel dispatches for various operations (preventing us from
// having to compile a version of nodes.cc with NVCC)

void saxpy_fast(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
    // Y <- A * X + Y
    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy_functor(A));
}

void set_to_value_of(int n, float* x0, float val)
{
    thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(x0);
    thrust::fill(thrust::device, dev_ptr, dev_ptr + n, val);
}

void set_to_value_of(int n, float* x0, float *val) {
    thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(x0);
    thrust::device_ptr<float> src_dev_ptr = thrust::device_pointer_cast(val);
    thrust::copy(src_dev_ptr, src_dev_ptr + n, dev_ptr);
}

void vpairwise_rank_loss(int n, float margin, const float* xgood, const float* xbad, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  binaryExprKernel<<<tb.first, tb.second>>>(n, xgood, xbad, y, FPairwiseRankLoss(margin));
}

void vpairwise_rank_loss_backward(int n, bool d_wrt_correct, const float* fx, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  if (d_wrt_correct) {
    accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FRectifyNegateBackward());
  } else {
    accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FRectifyBackward());
  }
}

void vcwise_product(int n, const float* x0, const float* x1, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  binaryExprKernel<<<tb.first, tb.second>>>(n, x0, x1, y, FProduct());
}

void vcwise_product_backward(int n, const float* dEdy, const float* x_other, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, dEdy, x_other, dEdx, FProduct());
}

void vcwise_quotient(int n, const float* x0, const float* x1, float* y) {
    auto tb = SizeToBlockThreadPair(n);
    binaryExprKernel << <tb.first, tb.second >> >(n, x0, x1, y, FQuotient());
}

void vcwise_quotient_backward(int n, const float* dEdy, const float* x_other, float* dEdx) {
    auto tb = SizeToBlockThreadPair(n);
    accBinaryExprKernel << <tb.first, tb.second >> >(n, dEdy, x_other, dEdx, FQuotient());
}

void vconstant_minusx(int n, float c, const float* x, float* y) {
    auto tb = SizeToBlockThreadPair(n);
    unaryExprKernel << <tb.first, tb.second >> >(n, x, y, FConstantMinus(c));
}

void vconstant_multiplyx(int n, float c, const float* x, float* y) {
    auto tb = SizeToBlockThreadPair(n);
    unaryExprKernel << <tb.first, tb.second >> >(n, x, y, FConstantMultiply(c));
}

void vconstant_multiplyx_backward(int n, float c, const float* x, float* y) {
    auto tb = SizeToBlockThreadPair(n);
    accUnaryExprKernel << <tb.first, tb.second >> >(n, x, y, FConstantMultiply(c));
}

void vexp(int n, const float* x, float* y) {
    auto tb = SizeToBlockThreadPair(n);
    unaryExprKernel << <tb.first, tb.second >> >(n, x, y, FExp());
}

void vnegate(int n, const float* x, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  unaryExprKernel<<<tb.first, tb.second>>>(n, x, y, FNegate());
}

void vnegate_backward(int n, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accUnaryExprKernel<<<tb.first, tb.second>>>(n, dEdf, dEdx, FNegate());
}

void vrelu(int n, const float* x, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  unaryExprKernel<<<tb.first, tb.second>>>(n, x, y, FRectify());
}

void vrelu_backward(int n, const float* fx, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FRectifyBackward());
}

void vtanh(int n, const float* x, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  unaryExprKernel<<<tb.first, tb.second>>>(n, x, y, FTanh());
}

void vtanh_backward(int n, const float* fx, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FTanhBackward());
}

void vlog(int n, const float* x, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  unaryExprKernel<<<tb.first, tb.second>>>(n, x, y, FLog());
}

void vlog_backward(int n, const float* fx, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FLogBackward());
}

void vlogistic(int n, const float* x, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  unaryExprKernel<<<tb.first, tb.second>>>(n, x, y, FLogisticSigmoid());
}

void vlogistic_backward(int n, const float* fx, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FLogisticSigmoidBackward());
}

void sqeucdist_backward(int n, const float* dEdy, const float* x0, const float* x1, float* dEdx, int i) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, x0, x1, dEdx, FEuclideanBackward(i, dEdy));
}

void sgd_update(int n, const float* g, float* x, float scale, float lambda) {
    auto tb = SizeToBlockThreadPair(n);
    accBinaryExprKernel << <tb.first, tb.second >> >(n, x, g, x, FL2SGDUpdate(lambda, scale));
}

void sgd_momentum_update(int n, const float* g, float* x, float* v, float scale, float lambda, float momentum) {
    auto tb = SizeToBlockThreadPair(n);
    accTripletExprKernel << <tb.first, tb.second >> >(n, x, g, v, x, FL2SGDMomentumUpdate(lambda, scale, momentum));
}

/** followed some examples of using thrust at
https://github.com/OrangeOwlSolutions/Thrust/blob/master/Calculating_the_norm_of_arrays.cu
*/
void rmsprop_momentum_update(int n, const float* g, float* x, float* v, float *r, float scale, float lambda, float momentum, float rho, float epsilon) {
    auto tb = SizeToBlockThreadPair(n);
    float squared_norm = thrust::transform_reduce(thrust::device_pointer_cast(g), thrust::device_pointer_cast(g + n), FSquare(), (float)0.0, thrust::plus<float>());
    *r = rho * (*r) + (1 - rho) * squared_norm;
    float den = sqrt(*r + epsilon);
    accTripletExprKernel << <tb.first, tb.second >> >(n, x, g, v, x, FL2SGDMomentumUpdate(lambda, scale / den, momentum));
    //CUDA_CHECK(hipFree(sqnorm));
}

void sqeucdist(int n, const float* x0, const float *x1, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  ker_sqeucdist<<<tb.first,tb.second>>>(n, x0, x1, y);
}

void l2_norm_reducer(int n, const float* x0, float* y, bool square, bool accumulate) {
  auto tb = SizeToBlockThreadPair(n);
  ker_l2_norm_reducer<<<tb.first,tb.second>>>(n, x0, y, square, accumulate);
}

void VectorSum(int rows, int cols, const float * a, float* c, const bool isColWise)
{
    assert(rows > 0 && cols > 0); // converting from size_t to int may cause overflow

    int m = cols;
    int n = rows;

    hipEvent_t done = nullptr;

    int blocksPerGrid = 0;
    if (isColWise) // col-wise
    {
        blocksPerGrid = (int)ceil(1.0 * m / MAX_THREADS_PER_BLOCK);
    }
    else
    {
        blocksPerGrid = (int)ceil(1.0 * n / MAX_THREADS_PER_BLOCK);
    }

    hipEventCreate(&done);
    _vectorSum<float> << <blocksPerGrid, MAX_THREADS_PER_BLOCK, 0, hipStreamDefault >> >(c, a, n, m, isColWise);
    hipEventRecord(done);
    hipEventSynchronize(done);
    hipEventDestroy(done);
}

/// assume that a is a vector with col dimension
void RowElementMultiplyWith(int arow, int acol, const float * a, int brow, int bcol, float * b)
{
    if (arow != 1 || acol != bcol)
    {
        abort();
    }

    int N = brow; 
    int M = acol;
    int blocksPerGrid = (int)ceil(1.0 * M / MAX_THREADS_PER_BLOCK);

    hipEvent_t done = nullptr;
    hipEventCreate(&done);
    _rowElementMultiplyWith<float> << <blocksPerGrid, MAX_THREADS_PER_BLOCK >> >(b, a, N, M);
    hipEventRecord(done);
    hipEventSynchronize(done);
    hipEventDestroy(done);
}

void logsoftmax(int row, int col, const float* x0, float* y) 
{
    hipStream_t t_stream = hipStreamDefault;

    int N = col;
    int M = row;
    hipEvent_t done = nullptr;
    hipEventCreate(&done);
    _assignColumnwiseLogSoftmaxOf<float> << <N, 512, 0, t_stream >> >(x0, y, N, M);
    
    hipEventRecord(done);
    
    hipEventSynchronize(done);
    
    hipEventDestroy(done);
}

void logsoftmax_backward(int row, int col, const float *fx, const float *dEdf, float *dEdx, float * gpu_softmax, float *grd)
{
    vexp(row * col, fx, gpu_softmax);
    VectorSum(row, col, dEdf, grd, true); 
    RowElementMultiplyWith(1, col, grd, row, col, gpu_softmax);

    auto tb = SizeToBlockThreadPair(col * row);
    accBinaryExprKernel << <tb.first, tb.second >> >(col * row, dEdf, gpu_softmax, dEdx, FSubtract());
}

void softmax(int row, int col, const float* x0, float* y)
{
    hipStream_t t_stream = hipStreamDefault;

    int N = col;
    int M = row;
    hipEvent_t done = nullptr;
    hipEventCreate(&done);
    _assignColumnwiseSoftmaxOf<float> << <N, MAX_THREADS_PER_BLOCK, 0, t_stream >> >(x0, y, N, M);

    hipEventRecord(done);

    hipEventSynchronize(done);

    hipEventDestroy(done);
}

/// see http://research.microsoft.com/pubs/226641/CNTKBook-20160121.pdf
void softmax_backward(int row, int col, const float *fx, const float *dEdf, float *dEdx, float *tmp_one_row)
{
    int n = row * col;
    auto tb = SizeToBlockThreadPair(n);
    float ods;
    ker_dotproduct << <tb.first, tb.second >> >(n, fx, dEdf, tmp_one_row);
    hipMemcpy(&ods, tmp_one_row, sizeof(float), hipMemcpyDeviceToHost);
    accBinaryExprKernel << <tb.first, tb.second >> >(n, fx, dEdf, dEdx, FSoftmaxBackward(-ods));
}

// adapted from NVIDIA example
__global__ void ker_pnlsoftmax(int n, int elem_idx, const float *x0, float* res, float* logz) {
  __shared__ float buf[256];
  for (int i = threadIdx.x; i < 256; i += blockDim.x) {
    float me = __int_as_float(0xff800000);
    for (int pos = i; pos < n; pos += 256) {
      const float d = x0[pos];
      me = d > me ? d : me;
    }
    buf[i] = me;
  }
  for (int stride = 128; stride > 0; stride >>= 1) {
    __syncthreads();
    for (int i = threadIdx.x; i < stride; i += blockDim.x)
        buf[i] = buf[i] > buf[stride + i] ? buf[i] : buf[stride + i];
  }
  __syncthreads();
  const float max_elem = buf[0];
  for (int i = threadIdx.x; i < 256; i += blockDim.x) {
    float sum = 0;
    for (int pos = i; pos < n; pos += 256)
      sum += expf(x0[pos] - max_elem);
    buf[i] = sum;
  }
  for (int stride = 128; stride > 0; stride >>= 1) {
    __syncthreads();
    for (int i = threadIdx.x; i < stride; i += blockDim.x)
        buf[i] += buf[stride + i];
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    float lz = log(buf[0]) + max_elem;
    logz[0] = lz;
    res[0] = lz - x0[elem_idx];
  }
}

void pnlsoftmax(int n, int elem_idx, const float* x0, float* y, float* logz) {
  auto tb = SizeToBlockThreadPair(n);
  ker_pnlsoftmax<<<tb.first,tb.second>>>(n, elem_idx, x0, y, logz);
}

__global__ void fixup_pnl(const float* dEdf, float* dEdxi, int i) {
  if (threadIdx.x == 0) dEdxi[i] -= dEdf[0];
}

void pnlsoftmax_backward(int n, int elem_idx, const float* x0, const float* dEdf, const float* logz, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accUnaryExprKernel<<<tb.first, tb.second>>>(n, x0, dEdx, FPtrNegLogSoftmaxBackward(logz, dEdf));
  fixup_pnl<<<1,1>>>(dEdf, dEdx, elem_idx);
}


void conv1dwide(const int n, const int m, const float* xs, const int k, const float *fx, float *fy)
{

    thrust::device_vector<float> dv((m + k) * n, 0.0);
    thrust::device_ptr<float> vp = dv.data();
    thrust::device_ptr<float> fp((float*)fx);
    thrust::device_ptr<float> xp((float*)xs);
    thrust::device_ptr<float> yp(fy);

    for (size_t tk = 0; tk < k; tk++)
    {
        for (size_t j = 0; j < m; j++)
            thrust::transform(xp + j * n, xp + (j + 1) * n, fp + tk * n, vp + tk * n + j * n, thrust::multiplies<float>());
    }
    thrust::copy(vp, vp + (m + k) * n, thrust::device_pointer_cast(fy));
}

void conv1dwide_backward(const int i, const int n, const int m, const float* xs, const int k, const float *fx, const float* dEdf, float *dEdx)
{
    thrust::device_vector<float> dv(m  * n, 0.0);
    thrust::device_ptr<float> vp = dv.data();
    thrust::device_ptr<float> fp((float*)fx);
    thrust::device_ptr<float> xp((float*)xs);
    thrust::device_ptr<float> d((float*)dEdf);
    thrust::device_ptr<float> yp(dEdx);

    for (size_t tk = 0; tk < k; tk++)
    {
        if (i == 0) { // derivative wrt input x
            for (size_t j = 0; j < m; j++)
                thrust::transform(d + j * n + tk*n, d + (j + 1) * n + tk*n, fp + tk * n, dv.data() + j * n, thrust::multiplies<float>());
        }
        else { // derivative wrt filter f
            for (size_t j = 0; j < m; j++)
                thrust::transform(d + j * n + tk*n, d + (j + 1) * n + tk*n, xp + j * n, dv.data() + tk * n, thrust::multiplies<float>());
        }
    }
    if (i == 0)
        thrust::transform(dv.data(), dv.data() + m * n, yp, yp, thrust::plus<float>());
    else 
        thrust::transform(dv.data(), dv.data() + k * n, yp, yp, thrust::plus<float>());
}

void addVectorToAllColumns(const int n, const float * xs, const int m, const float* fx, float *fy)
{
    thrust::device_ptr<float> fp((float*)fx);
    thrust::device_ptr<float> xp((float*)xs);
    thrust::device_ptr<float> yp(fy);
    for (size_t j = 0; j < n / m; j++)
        thrust::transform(xp + j * m, xp + (j + 1) * m, fp, yp + j * m, thrust::plus<float>());
}

void addVectorToAllColumns_backward(const int i, const int r, const int c, const float* dEdf, float *dEdxi)
{
    thrust::device_ptr<const float> dp(dEdf);
    thrust::device_ptr<float> dx(dEdxi);

    if (i == 0)
    {
        // x
        thrust::transform(dp, dp + r * c, dx, dx, thrust::plus<float>());
    }
    else
    {
        // bias
        for (int k = 0; k < c; k++)
            thrust::transform(dp + k * r, dp + (k + 1)*r, dx, dx, thrust::plus<float>());
    }
}

/**
stride : the jump step
*/
void foldRows(const int n, const int m, const float *xs, const int stride, const int orows, float *fy)
{
    thrust::device_ptr<float> xp((float*)xs), pp;
    thrust::device_ptr<float> yp(fy);
    thrust::host_vector<float> vo(orows * m);

    pp = xp;
    for (size_t j = 0; j < m; j++)
    {
        for (size_t r = 0; r < orows; r++)
        {
            vo[j * orows + r] = thrust::reduce(pp, pp + stride);
            pp += stride;
        }
    }
}

void foldRows_backward(const int orows, const float* dEdf, const int n, const int m, float *fy)
{
    thrust::device_ptr<float> dp((float*)dEdf);
    thrust::device_ptr<float> yp(fy);

    for (int i = 0; i < orows; ++i)
    {
        int stride = n / orows;
        for (int j = 0; j < m; j++)
        { // loop over columns
            for (int k = 0; k < stride; k++)
            {
                *(yp + i * stride + k + j * n) += *(dp + i + j * n);
            }
        }
    }
}

void kMaxPooling(const int n, const int m, const float *xs, const int k, float *fy, int* aux_mem)
{
    thrust::device_ptr<float> xp((float*)xs), pp;
    thrust::device_ptr<float> yp(fy);
    thrust::device_vector<float> vo(m);
    thrust::device_vector<float> vp(k);

    pp = xp;

    int* maxmap = static_cast<int*>(aux_mem);
    size_t mi = 0;
    for (unsigned i = 0; i < n; ++i) {
        for (size_t j = 0; j < m; j++)
            vo[j] = (*(pp + i + j * n));
        thrust::sort(thrust::device, vo.data(), vo.data() + m);

        size_t mk = 0;
        for (int j = 0; j < m; j++)
        {
            if (mk == k)
                break;
            if (*(pp + i + j * n) >= vo[m - k])
            {
                *(yp + i + mk * n) = *(pp + i + j*n);
                hipMemcpy(&maxmap[mi], &j, sizeof(int), hipMemcpyHostToDevice); 
                mi++;
                mk++;
            }
        }
    }
}

void kMaxPooling_backward(const int n, const int m, const float *xs, const int k, const float * dEdf, float *dEdxi, const int* aux_mem)
{
    const int* maxmap = aux_mem;
    int mk = 0;
    int oj;
    thrust::device_ptr<const float> xp(xs);
    thrust::device_ptr<const float> dp(dEdf);
    thrust::device_ptr<float> yp(dEdxi);
    thrust::host_vector<int> hv(n, 0);
    hipMemcpy(hv.data(), maxmap, sizeof(int)*n, hipMemcpyDeviceToHost);

    for (unsigned i = 0; i < n; ++i) {
        for (unsigned j = 0; j < k; ++j) {
            oj = hv[mk++];
            if (oj < k && oj >= 0){
                thrust::transform(dp + i + j * n, dp + i + j * n + 1, yp + i + oj * n, yp + i + oj * n, thrust::plus<float>());
            }
        }
    }
}


} // namespace gpu
} // namespace cnn
